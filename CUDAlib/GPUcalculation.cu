#include "GPUcalculation.h"

GPUcalculation::GPUcalculation()
{
    checkCublas(hipblasCreate(&handle), "Failed to create cuBLAS handle");
}

GPUcalculation::~GPUcalculation()
{
    hipblasDestroy(handle);
}

void GPUcalculation::getMatrixMultiply(const Eigen::MatrixXd &A, const Eigen::MatrixXd &B)
{
    int m = A.rows();
    int n = B.cols();
    int k = A.cols();
    Eigen::MatrixXd C(m, n);

    const double alpha = 1.0;
    const double beta = 0.0;

    size_t size_of_A = m * k * sizeof(double);
    size_t size_of_B = n * k * sizeof(double);
    size_t size_of_C = m * n * sizeof(double);

    double *p_A = nullptr, *p_B = nullptr, *p_C = nullptr;

    checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
    checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");
    checkCuda(hipMalloc((void **)&p_C, size_of_C), "Failed to allocate GPU memory for C");

    checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");
    checkCuda(hipMemcpy(p_B, B.data(), size_of_B, hipMemcpyHostToDevice), "Failed to copy data to GPU");

    checkCublas(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            m, n, k,
                            &alpha,
                            p_A, m,
                            p_B, k,
                            &beta,
                            p_C, m),
                "Failed to perform DGEMM");

    checkCuda(hipMemcpy(C.data(), p_C, size_of_C, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

    checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
    checkCuda(hipFree(p_B), "Failed to free GPU memory for B");
    checkCuda(hipFree(p_C), "Failed to free GPU memory for C");
}

Eigen::MatrixXd GPUcalculation::getMatrixTranspose(const Eigen::MatrixXd &A)
{
    int rows = A.rows();
    int cols = A.cols();
    Eigen::MatrixXd B(cols, rows);

    size_t size_of_A = rows * cols * sizeof(double);
    size_t size_of_B = rows * cols * sizeof(double);

    double *p_A = nullptr, *p_B = nullptr;

    checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
    checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");

    checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");

    const double alpha = 1.0;
    const double beta = 0.0;

    checkCublas(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                            cols, rows,
                            &alpha,
                            p_A, rows,
                            &beta,
                            p_A, cols,
                            p_B, cols),
                "Failed to perform matrix transpose");

    checkCuda(hipMemcpy(B.data(), p_B, size_of_B, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

    checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
    checkCuda(hipFree(p_B), "Failed to free GPU memory for B");

    return B;
}

void GPUcalculation::checkCuda(hipError_t result, const char *msg)
{
    if (result != hipSuccess)
    {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void GPUcalculation::checkCublas(hipblasStatus_t result, const char *msg)
{
    if (result != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "cuBLAS Error: " << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}
