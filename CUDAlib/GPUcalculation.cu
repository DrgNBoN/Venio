#include "GPUcalculation.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

inline void checkCuda(hipError_t result, const char *msg)
{
    if (result != hipSuccess)
    {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

inline void checkCublas(hipblasStatus_t result, const char *msg)
{
    if (result != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "cuBLAS Error: " << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}

inline hipblasHandle_t getCublasHandle(void* handle) noexcept
{
    return reinterpret_cast<hipblasHandle_t>(handle);
}

GPUcalculation::GPUcalculation()
{
    hipblasHandle_t tHandle;
    checkCublas(hipblasCreate(&tHandle), "Failed to create cuBLAS handle");
    handle = static_cast<internal_handle>(tHandle);
}

GPUcalculation::~GPUcalculation()
{
    hipblasDestroy(getCublasHandle(handle));
}

void GPUcalculation::getMatrixMultiply(const Eigen::MatrixXd &A, const Eigen::MatrixXd &B)
{
    int m = static_cast<int>(A.rows());
    int n = static_cast<int>(B.cols());
    int k = static_cast<int>(A.cols());
    Eigen::MatrixXd C(m, n);

    const double alpha = 1.0;
    const double beta = 0.0;

    size_t size_of_A = m * k * sizeof(double);
    size_t size_of_B = n * k * sizeof(double);
    size_t size_of_C = m * n * sizeof(double);

    double *p_A = nullptr, *p_B = nullptr, *p_C = nullptr;

    checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
    checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");
    checkCuda(hipMalloc((void **)&p_C, size_of_C), "Failed to allocate GPU memory for C");

    checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");
    checkCuda(hipMemcpy(p_B, B.data(), size_of_B, hipMemcpyHostToDevice), "Failed to copy data to GPU");

    checkCublas(hipblasDgemm(getCublasHandle(handle),
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            m, n, k,
                            &alpha,
                            p_A, m,
                            p_B, k,
                            &beta,
                            p_C, m),
                "Failed to perform DGEMM");

    checkCuda(hipMemcpy(C.data(), p_C, size_of_C, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

    checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
    checkCuda(hipFree(p_B), "Failed to free GPU memory for B");
    checkCuda(hipFree(p_C), "Failed to free GPU memory for C");
}

Eigen::MatrixXd GPUcalculation::getMatrixTranspose(const Eigen::MatrixXd &A)
{
    int rows = static_cast<int>(A.rows());
    int cols = static_cast<int>(A.cols());
    Eigen::MatrixXd B(cols, rows);

    size_t size_of_A = rows * cols * sizeof(double);
    size_t size_of_B = rows * cols * sizeof(double);

    double *p_A = nullptr, *p_B = nullptr;

    checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
    checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");

    checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");

    const double alpha = 1.0;
    const double beta = 0.0;

    checkCublas(hipblasDgeam(getCublasHandle(handle),
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            cols, rows,
                            &alpha,
                            p_A, rows,
                            &beta,
                            p_A, cols,
                            p_B, cols),
                "Failed to perform matrix transpose");

    checkCuda(hipMemcpy(B.data(), p_B, size_of_B, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

    checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
    checkCuda(hipFree(p_B), "Failed to free GPU memory for B");

    return B;
}
