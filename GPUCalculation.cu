#include <hipblas.h>
#include <hip/hip_runtime.h>

class GPUcalculation
{
public:
    GPUcalculation()
    {
        checkCublas(hipblasCreate(&handle), "Failed to create cuBLAS handle");
    }

    ~GPUcalculation()
    {
        hipblasDestroy(handle);
    }

    void getMatrixMultiply(const Eigen::MatrixXd &A, const Eigen::MatrixXd &B)
    {
        int m = A.rows();
        int n = B.cols();
        int k = A.cols();
        Matrixd C(m, n);

        const double alpha = 1.0;
        const double beta = 0.0;

        size_t size_of_A = m * k * sizeof(double);
        size_t size_of_B = n * k * sizeof(double);
        size_t size_of_C = m * n * sizeof(double);

        double *p_A = nullptr, *p_B = nullptr, *p_C = nullptr;

        checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
        checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");
        checkCuda(hipMalloc((void **)&p_C, size_of_C), "Failed to allocate GPU memory for C");

        checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");
        checkCuda(hipMemcpy(p_B, B.data(), size_of_B, hipMemcpyHostToDevice), "Failed to copy data to GPU");

        checkCublas(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                m, n, k,
                                &alpha,
                                p_A, m,
                                p_B, k,
                                &beta,
                                p_C, m),
                    "Failed to perform DGEMM");

        checkCuda(hipMemcpy(C.data(), p_C, size_of_C, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

        checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
        checkCuda(hipFree(p_B), "Failed to free GPU memory for B");
        checkCuda(hipFree(p_C), "Failed to free GPU memory for C");
    }
    Matrixd getMatrixTranspose(const Eigen::MatrixXd &A)
    {
        int rows = A.rows();
        int cols = A.cols();
        Matrixd B(cols, rows);

        size_t size_of_A = rows * cols * sizeof(double);
        size_t size_of_B = rows * cols * sizeof(double);

        double *p_A = nullptr, *p_B = nullptr;

        checkCuda(hipMalloc((void **)&p_A, size_of_A), "Failed to allocate GPU memory for A");
        checkCuda(hipMalloc((void **)&p_B, size_of_B), "Failed to allocate GPU memory for B");

        checkCuda(hipMemcpy(p_A, A.data(), size_of_A, hipMemcpyHostToDevice), "Failed to copy data to GPU");

        const double alpha = 1.0;
        const double beta = 0.0;

        checkCublas(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                cols, rows,
                                &alpha,
                                p_A, rows,
                                &beta,
                                p_A, cols,
                                p_B, cols),
                    "Failed to perform matrix transpose");

        checkCuda(hipMemcpy(B.data(), p_B, size_of_B, hipMemcpyDeviceToHost), "Failed to copy data from GPU");

        checkCuda(hipFree(p_A), "Failed to free GPU memory for A");
        checkCuda(hipFree(p_B), "Failed to free GPU memory for B");

        return B;
    }

private:
    hipblasHandle_t handle;
    void checkCuda(hipError_t result, const char *msg)
    {
        if (result != hipSuccess)
        {
            std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    void checkCublas(hipblasStatus_t result, const char *msg)
    {
        if (result != HIPBLAS_STATUS_SUCCESS)
        {
            std::cerr << "cuBLAS Error: " << msg << std::endl;
            exit(EXIT_FAILURE);
        }
    }
};